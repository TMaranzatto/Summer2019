#include "hip/hip_runtime.h"
/*

Copyright 2012-2013 Indian Institute of Technology Kanpur. All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer.
2. Redistributions in binary form must reproduce the above copyright notice,
this list of conditions, and the following disclaimer in the documentation
and/or other materials provided with the distribution.

THIS SOFTWARE IS PROVIDED BY INDIAN INSTITUTE OF TECHNOLOGY KANPUR ``AS IS''
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL INDIAN INSTITUTE OF TECHNOLOGY KANPUR OR
THE CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT
OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
POSSIBILITY OF SUCH DAMAGE.

The views and conclusions contained in the software and documentation are
those of the authors and should not be interpreted as representing official
policies, either expressed or implied, of Indian Institute of Technology Kanpur.

*/

/**********************************************************************************

 Lock-free linked list for CUDA; tested for CUDA 4.2 on 32-bit Ubuntu 10.10 and 64-bit Ubuntu 12.04.
 Developed at IIT Kanpur.

 Inputs: Percentage of add and delete operations (e.g., 30 50 for 30% add and 50% delete)
 Output: Prints the total time (in milliseconds) to execute the the sequence of operations

 Compilation flags: -O3 -arch sm_20 -I ~/NVIDIA_GPU_Computing_SDK/C/common/inc/ -DNUM_ITEMS=num_ops -DFACTOR=num_ops_per_thread -DKEYS=num_keys

 NUM_ITEMS is the total number of operations (mix of add, delete, search) to execute.

 FACTOR is the number of operations per thread.

 KEYS is the number of integer keys assumed in the range [10, 9+KEYS].
 The paper cited below states that the key range is [0, KEYS-1]. However, we have shifted the range by +10 so that
 the head sentinel key (the minimum key) can be chosen as zero. Any positive shift other than +10 would also work.

 The include path ~/NVIDIA_GPU_Computing_SDK/C/common/inc/ is needed for cutil.h.

 Related work:

 Prabhakar Misra and Mainak Chaudhuri. Performance Evaluation of Concurrent Lock-free Data Structures
 on GPUs. In Proceedings of the 18th IEEE International Conference on Parallel and Distributed Systems,
 December 2012.

***************************************************************************************/

#include"cutil.h"		// Comment this if cutil.h is not available
#include"hip/hip_runtime.h"
#include"stdio.h"

#if __WORDSIZE == 64
typedef unsigned long long LL;
#else
typedef unsigned int LL;
#endif

// Number of threads per block
#define NUM_THREADS 64

// Supported operations
#define ADD (0)
#define DELETE (1)
#define SEARCH (2)

// Definition of generic node class

class __attribute__((aligned (16))) Node
{
  public:
    LL key;
    LL next;

    // Create a next field from a reference and mark bit
    __device__ __host__ LL CreateRef(Node* ref, bool mark)
    {
      LL val=(LL)ref;
      val=val|mark;
      return val;
    }

    __device__ __host__ void SetRef(Node* ref, bool mark)
    {
      next=CreateRef(ref, mark);
    }

    // Extract the reference from a next field
    __device__ Node* GetReference()
    {
      LL ref=next;
      return (Node*)((ref>>1)<<1);
    }

    // Extract the reference and mark bit from a next field
    __device__ Node* Get(bool* marked)
    {
      marked[0]=next%2;
      return (Node*)((next>>1)<<1);
    }

    // CompareAndSet wrapper
    __device__ bool CompareAndSet(Node* expectedRef, Node* newRef, bool oldMark, bool newMark)
    {
      LL oldVal = (LL)expectedRef|oldMark;
      LL newVal = (LL)newRef|newMark;
      LL oldValOut=atomicCAS(&(next), oldVal, newVal);
      if (oldValOut==oldVal) return true;
      return false;
    }

    // Constructor for sentinel nodes
    Node(LL k)
    {
      key=k;
      next=CreateRef((Node*)NULL,false);
    }
};

__device__ Node** nodes;			// Pool of pre-allocated nodes
__device__ unsigned int pointerIndex=0;		// Index into pool of free nodes

// Function for creating a new node when requested by an add operation

__device__ Node* GetNewNode(LL key)
{
  LL ind=atomicInc(&pointerIndex, NUM_ITEMS);
  Node* n=nodes[ind];
  n->key=key;
  n->SetRef(NULL, false);
  return n;
}

// Window of node containing a particular key

class Window
{
  public:
    Node* pred;		// Predecessor of node holding the key being searched
    Node* curr;		// The node holding the key being searched (if present)

    __device__ Window(Node* myPred, Node* myCurr)
    {
      pred=myPred;
      curr=myCurr;
    }
};

// Lock-free linked list

class LinkedList
{
  public:
    __device__ void Find(Window*, LL);			// Helping method
    __device__ bool Add(LL);
    __device__ bool Delete(LL);
    __device__ bool Search(LL);

    Node* head;
    Node* tail;

    LinkedList()
    {
        Node* h=new Node(0);				// Head sentinel
#if __WORDSIZE == 64
        Node* t=new Node((LL)0xffffffffffffffff);           // Tail sentinel
#else
        Node* t=new Node((LL)0xffffffff);			// Tail sentinel
#endif

#ifdef _CUTIL_H_
        CUDA_SAFE_CALL(hipMalloc((void**)&head, sizeof(Node)));
#else
        hipMalloc((void**)&head, sizeof(Node));
#endif

#ifdef _CUTIL_H_
        CUDA_SAFE_CALL(hipMalloc((void**)&tail, sizeof(Node)));
#else
        hipMalloc((void**)&tail, sizeof(Node));
#endif
        h->next=(LL)tail;
#ifdef _CUTIL_H_
        CUDA_SAFE_CALL(hipMemcpy(head, h, sizeof(Node), hipMemcpyHostToDevice));
#else
        hipMemcpy(head, h, sizeof(Node), hipMemcpyHostToDevice);
#endif

#ifdef _CUTIL_H_
        CUDA_SAFE_CALL(hipMemcpy(tail, t, sizeof(Node), hipMemcpyHostToDevice));
#else
        hipMemcpy(tail, t, sizeof(Node), hipMemcpyHostToDevice);
#endif
      }
};

// Find the window holding key
// On the way clean up logically deleted nodes (those with set marked bit)

__device__ void
LinkedList::Find(Window* w, LL key)
{
  Node* pred;
  Node* curr;
  Node* succ;
  bool marked[]={false};
  bool snip;

  retry:
  while(true){
     pred=head;
     curr=pred->GetReference();
     while(true){
        succ=curr->Get(marked);
        while(marked[0]){
           snip=pred->CompareAndSet(curr, succ, false, false);
           if(!snip) goto retry;
	   curr=succ;
	   succ=curr->Get(marked);
        }
        if(curr->key >= key){
           w->pred=pred;
           w->curr=curr;
           return;
	}
	pred=curr;
	curr=succ;
     }
  }
}

__device__ bool 
LinkedList::Search(LL key)
{
  bool marked;
  Node* curr = head;
  while(curr->key<key){
     curr=curr->GetReference();
     Node* succ = curr->Get(&marked);
  }
  return((curr->key == key) && !marked);
}
   
__device__ bool
LinkedList::Delete(LL key)
{
  Window w(NULL, NULL);
  bool snip;
  while(true){
     Find(&w, key);
     Node* curr=w.curr;
     Node* pred=w.pred;
     if(curr->key!=key){
        return false;
     }
     else{
        Node* succ = curr->GetReference();
        snip=curr->CompareAndSet(succ, succ, false, true);
	if(!snip) continue;
	pred->CompareAndSet(curr, succ, false, false);
	return true;
     }
  }
}

__device__ bool
LinkedList::Add(LL key)
{
  Node* pointer=GetNewNode(key);
  Window w(NULL, NULL);
  while(true){
     Find(&w, key);
     Node* pred=w.pred;
     Node* curr=w.curr;
     if (curr->key == key) return false;
     pointer->key=key;
     pointer->SetRef(curr, false);
     bool test=(pred->CompareAndSet(curr, pointer, false, false));
     if(test) return true;
  }
}

__device__ LinkedList* list;            // The linked list

// Kernel for initializing device memory

__global__ void init(LinkedList* List)
{
  list=List;
}

// The main kernel

__global__ void kernel(LL* items, LL* op, LL* result, Node** n)
{
  // The array items holds the sequence of keys
  // The array op holds the sequence of operations
  // The array result, at the end, will hold the outcome of the operations
  // n points to an array of pre-allocated free linked list nodes

  nodes=n;
  int tid;
  int i;
  for(i=0;i<FACTOR;i++){    		// FACTOR is the number of operations per thread
    tid=i*gridDim.x*blockDim.x+blockIdx.x*blockDim.x+threadIdx.x;
    if(tid>=NUM_ITEMS) return;

    // Grab the operation and the associated key and execute
    LL itm=items[tid];
    if(op[tid]==ADD){
      result[tid]=list->Add(itm);
    }
    if(op[tid]==DELETE){
      result[tid]=list->Delete(itm);
    }
    if(op[tid]==SEARCH){
      result[tid]=list->Search(itm);
    }
  }
}

int main(int argc, char** argv)
{
  if (argc != 3) {
     printf("Need two arguments: percent add ops and percent delete ops (e.g., 30 50 for 30%% add and 50%% delete).\nAborting...\n");
     exit(1);
  }

  int adds=atoi(argv[1]);
  int deletes=atoi(argv[2]);

   if (adds+deletes > 100) {
     printf("Sum of add and delete precentages exceeds 100.\nAborting...\n");
     exit(1);
  }

  // Allocate linked list

  LinkedList* list=new LinkedList();
  LinkedList* Clist;
  int i;
#ifdef _CUTIL_H_
  CUDA_SAFE_CALL(hipMalloc((void**)&Clist, sizeof(LinkedList)));
#else
  hipMalloc((void**)&Clist, sizeof(LinkedList));
#endif

#ifdef _CUTIL_H_
  CUDA_SAFE_CALL(hipMemcpy(Clist, list, sizeof(LinkedList), hipMemcpyHostToDevice));
#else
  hipMemcpy(Clist, list, sizeof(LinkedList), hipMemcpyHostToDevice);
#endif

  // Initialize the device memory
  init<<<1, 32>>>(Clist);

  LL op[NUM_ITEMS];		// Array of operations
  LL items[NUM_ITEMS];		// Array of keys associated with operations
  LL result[NUM_ITEMS];		// Array of outcomes

  srand(0);

  // NUM_ITEMS is the total number of operations to execute
  for(i=0;i<NUM_ITEMS;i++){
    items[i]=10+rand()%KEYS;	// Keys
  }

  // Populate the op sequence
  for(i=0;i<(NUM_ITEMS*adds)/100;i++){
    op[i]=ADD;
  }
  for(;i<(NUM_ITEMS*(adds+deletes))/100;i++){
    op[i]=DELETE;
  }
  for(;i<NUM_ITEMS;i++){
    op[i]=SEARCH;
  }

  adds=(NUM_ITEMS*adds)/100;

  // Allocate device memory

  LL* Citems;
  LL* Cop;
  LL* Cresult;
#ifdef _CUTIL_H_
  CUDA_SAFE_CALL(hipMalloc((void**)&Cresult, sizeof(LL)*NUM_ITEMS));
  CUDA_SAFE_CALL(hipMalloc((void**)&Citems, sizeof(LL)*NUM_ITEMS));
  CUDA_SAFE_CALL(hipMalloc((void**)&Cop, sizeof(LL)*NUM_ITEMS));
  CUDA_SAFE_CALL(hipMemcpy(Citems,items, sizeof(LL)*NUM_ITEMS, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(Cop, op, sizeof(LL)*NUM_ITEMS, hipMemcpyHostToDevice));
#else
  hipMalloc((void**)&Cresult, sizeof(LL)*NUM_ITEMS);
  hipMalloc((void**)&Citems, sizeof(LL)*NUM_ITEMS);
  hipMalloc((void**)&Cop, sizeof(LL)*NUM_ITEMS);
  hipMemcpy(Citems,items, sizeof(LL)*NUM_ITEMS, hipMemcpyHostToDevice);
  hipMemcpy(Cop, op, sizeof(LL)*NUM_ITEMS, hipMemcpyHostToDevice);
#endif

  Node* pointers[adds];
  Node** Cpointers;

  // Allocate the pool of free nodes

  for(i=0;i<adds;i++){
#ifdef _CUTIL_H_
    CUDA_SAFE_CALL(hipMalloc((void**)&pointers[i], sizeof(Node)));
#else
    hipMalloc((void**)&pointers[i],sizeof(Node));
#endif
  }
  
#ifdef _CUTIL_H_
  CUDA_SAFE_CALL(hipMalloc((void**)&Cpointers, sizeof(Node*)*adds));
  CUDA_SAFE_CALL(hipMemcpy(Cpointers, pointers, sizeof(Node*)*adds, hipMemcpyHostToDevice));
#else
  hipMalloc((void**)&Cpointers, sizeof(Node*)*adds);
  hipMemcpy(Cpointers, pointers, sizeof(Node*)*adds, hipMemcpyHostToDevice);
#endif

  // Calculate the number of thread blocks
  // NUM_ITEMS = total number of operations to execute
  // NUM_THREADS = number of threads per block
  // FACTOR = number of operations per thread

  int blocks=(NUM_ITEMS%(NUM_THREADS*FACTOR)==0)?NUM_ITEMS/(NUM_THREADS*FACTOR):(NUM_ITEMS/(NUM_THREADS*FACTOR))+1;

  // Launch main kernel

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
  
  kernel<<<blocks, NUM_THREADS>>>(Citems, Cop, Cresult, Cpointers);
  
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float time;
  hipEventElapsedTime(&time, start, stop);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  // Print kernel execution time in milliseconds

  printf("%lf\n",time);

  // Check for errors

  hipError_t error= hipGetLastError();
  if(hipSuccess!=error){
    printf("error:CUDA ERROR (%d) {%s}\n",error,hipGetErrorString(error));
    exit(-1);
  }

  // Move results back to host memory

#ifdef _CUTIL_H_
  CUDA_SAFE_CALL(hipMemcpy(result, Cresult, sizeof(LL)*NUM_ITEMS, hipMemcpyDeviceToHost));
#else
  hipMemcpy(result, Cresult, sizeof(LL)*NUM_ITEMS, hipMemcpyDeviceToHost);
#endif

  return 0;
}
