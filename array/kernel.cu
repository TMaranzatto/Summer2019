#include "hip/hip_runtime.h"
#include <cstddef>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <algorithm>

struct arrayNode {
	int array[64];
	unsigned long long int bitmap;
	int min;
	int max;
	arrayNode* next;
	int seqLock;
	bool isStart;

	arrayNode(int minn, int maxx, arrayNode* nextt){
		bitmap = 0;
		min = minn;
		max = maxx;
		next = nextt;
		seqLock = 0;
		isStart = false;
	}
	~arrayNode(){}
};

//this implementation assumes a uniform distro of keys
//as well as naiive partitions
//no merging implemented yet
//assume that our value is within some reasonable range
__global__
void insert(arrayNode *start, int value){
	startInsert:
	//finding the location to insert
	arrayNode *current = start;
	//while we are not in the 
	while ((current->next)->next != NULL) {
		//1. find location to insert into, and skip the first dummy node
		if (current->min <= value && current->max >= value && current->isStart == true) {
			int insertion_location = (int)(value / ((current->max - current->min) / 64));
			int dir = -1;
			bool useCurrDir = false;
			bool successful_insertion_flag = false;

		//2. try to insert at the location in the array the value
		//would ideally exist in
			for (int i = 0; i < 64; i++) {
				int search_location = insertion_location + dir * i;
				insertion_location = search_location;
				unsigned long long int bitmap_location = 1 << search_location;

				//checking if we are outside bounds
				if (search_location < 0 || search_location >= 64) { break; }

				//checking if we are above the target value
				//which would break the sorting situation

				if (dir == -1) {
					if (current->array[search_location] > value && useCurrDir == false) { 
						dir *= -1; 
						useCurrDir = true; 
						continue;
					}
					else if (current->array[search_location] > value && useCurrDir == true){
						break;
					}
				}

				//and checking for below target
				else{
					if (current->array[search_location] < value && useCurrDir == false) { 
					dir *= -1; 
					useCurrDir = true; 
					continue;
					}
					else if (current->array[search_location] < value && useCurrDir == true){
						break;
					}
				}

				//if we succeed all the above value and boundary conditions
				//then we try to modify data in the array through the proxy
				//of our bitmap
				unsigned long long int previousValue = atomicOr(current->bitmap, bitmap_location);
				if ((previousValue >> bitmap_location) & 1 == 0) {
					current->array[search_location] = value;
					//setting flag for safety.
					//could get rid of this in refactoring
					successful_insertion_flag = true;
					return;
				}

				//otherwise, lets loop again and hope it works
				else {
					if( useCurrDir == false){ dir *= -1; }
					continue;
				}
			}

		//3. if above fails, do the split routine
			if (successful_insertion_flag == false) {
				if(current->seqLock%2 == 1){
					goto startInsert;
				}
				atomicCAS(current->seqLock)
				//NEED LOCK HERE
				arrayNode *new_arrayNode = new arrayNode(-1,-1, NULL);
				//very slow here but should be working
				//need to speed this up later

				//setting new array values
				int minval = 10000000;
				int maxval = 0;
				for (int i = 0; i < 32; i++) {
					int new_value = current->array[32 + i]
					if(new_value != 0){
						new_arrayNode->array[2 * i] = new_value;
						new_arrayNode->bitmap |= unsigned long long int (1<<(2 * i));
					}
					minval = min(minval, new_value)
					maxval = max(minval, cnew_value)
					//new_arrayNode->array[(2 * i) + 1] = current->array[32 + i];
				}
				new_arrayNode->min = minval;
				new_arrayNode->max = maxval;

				//and old array values
				minval = maxval + 1;
				maxval = 0;
				for (int i = 31; i >= 0; i--) {
					int new_value = current->array[i]
					if(new_value != 0){
						new_arrayNode->array[2 * i + 1] = new_value;
						new_arrayNode->bitmap |= unsigned long long int (1<<(2 * i + 1));
					}
					current->array[2 * i + 1] = current->array[i];
					maxval = max(minval, current->array[i])
				}
				current->min = minval;
				current->max = maxval;
				
				//finally setting the node pointers 
				new_arrayNode->next = current->next;
				current->next = new_arrayNode;
			}
		}
		//TODO
		//4. else, if this array and its right neighbor are < third full
		//merge them
		
		else {
			current = current->next;
			continue;
		}
	}

}

//returns "random" value for threadId%64 if not taken
//else returns NULL
//assumes array is for integers for simplicity
//can template this later
__global__
void get(int* array, unsigned long long int* bitmap) {
	printf("your thread is %d.%d.\n", blockIdx.x * blockDim.x, threadIdx.x);

	const int resolution = 8;
	int jumps[resolution] = { 3, 5, 7, 11, 13, 17, 19, 23 };
	//Loop through all 64 elements to see if we find one that works
	//hopefully our thdId gives us a result immediatly
	for(int k = 0; k < 64; k++){
		int flag = 0;
		int test = 0;
		//checking edge case that the array is empty
		//send a message to host
		if (*bitmap == ULLONG_MAX) {
			//do something
			
			flag = -1;
			#if __CUDA_ARCH__ >= 200
				printf("%d\n", flag);
			#endif
		}
		int jump = jumps[threadIdx.x % resolution];
		unsigned long long int i = (blockIdx.x * blockDim.x + threadIdx.x + jump) % 64;
		unsigned long long int loc = 1 << i;

		unsigned long long int previousValue = atomicOr(bitmap, loc);
		if ((previousValue >> i) & 1 == 0) {
			//do something with the value
			flag = 1;
			#if __CUDA_ARCH__ >= 200
						printf("%d\n", flag);
			#endif
		}

		else{
			//else try next element
			continue;
			#if __CUDA_ARCH__ >= 200
				printf("try again\n");
			#endif
		}

	}
}

int main(void) {
	unsigned long long int *bitmap;
	int *arr;

	hipMallocManaged(&bitmap, sizeof(unsigned long long int));
	hipMallocManaged(&arr, 64*sizeof(int));

	//dumb init of array
	for (int i = 0; i < 64; i++) {
		arr[i] = i;
	}
	printf("processing...\n");

	get<<<1, 16>>>(arr, bitmap);
	hipDeviceSynchronize();

	printf("task complete.\n");
	hipFree(bitmap);
	hipFree(arr);


}